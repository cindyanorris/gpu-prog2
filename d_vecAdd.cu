#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "d_vecAdd.h"

//use this as the size of your blocks (number of threads per block)
#define BLOCKDIM 512 

__global__ void d_vecAddKernel(float * d_A, float * d_B, float * d_C, int n);

/*  d_vecAdd
    Performs the vector add on the GPU (the device).
    A and B are pointers to two vectors to add together.
    The result is stored in the vector pointed to by C.
    n is the length of the vectors.

    returns the amount of time it takes to perform the
    vector add 
*/
float d_vecAdd(float* A, float* B, float* C, int n)
{
    float gpuMsecTime;
    hipEvent_t start_gpu, stop_gpu;

    //time the sum of the two vectors
    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));
    CHECK(hipEventRecord(start_gpu));

    //missing code goes here
    //1) create vectors on the device
    //2) copy A and B vectors into device vectors
    //3) launch the kernel
    //4) wait for the kernel threads to complete
    //5) copy the result vector into the C vector
    //6) free space allocated for vectors on the device
    //Don't forget to use the CHECK macro on your cuda calls

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));
    return gpuMsecTime;
}

/*  
    d_vecAddKernel
    This function contains the kernel code. This code will be
    executed by every thread created by the kernel launch.
    d_A and d_B are pointers to two vectors on the device to add together.
    The result is stored in the vector pointed to by d_C.
    n is the length of the vectors.
*/
__global__ void d_vecAddKernel(float * d_A, float * d_B, float * d_C, int n)
{
    //add the missing body
}      

