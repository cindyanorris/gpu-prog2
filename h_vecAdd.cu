#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "h_vecAdd.h"

/* h_vecAdd
   Performs the vector add on the CPU (the host).
   A and B are pointers to two vectors to add together.
   The result is stored in the vector pointed to by C.
   n is the length of the vectors.

   returns the amount of time it takes to perform the
   vector add
*/
float h_vecAdd(float* A, float* B, float* C, int n)
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_cpu));  
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));   

    int i;
    for (i = 0; i < n; i++)
    {
        C[i] = A[i] + B[i];
    }
   
    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu)); 
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

